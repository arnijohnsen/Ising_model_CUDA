#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Lattice dimensions and thread_num
#define thread_num 512 // Should be 2^k
#define grid_dim_x 1024
#define grid_size 1048576
#define iter 500
#define iterbal 100

void calc_cpu(float B, float kT, float Q, float *E_avg, float *M_avg, float *E_var, float *M_var);
__global__ void set_lattice(bool *lattice);
__global__ void iterate_grid(float B, float kT, float Q, bool round, bool *dev_lattice, float *d_E_vec, float *d_M_vec, int seed);
__global__ void reset_vec(float *vec);
__global__ void vec_sum(float *vec, float *result);
__global__ void set_val(float *variable, float value);
__global__ void add_val(float *variable, float *addition);
__device__ int posMod(int number, int modulus);
__device__ int indexMap(int xi, int yi);

// Inn í Accumulator eru lesin gildi í úrtaki, haldið er utan um meðalgildi og 
// dreifni þess úrtaks. 
class Accumulator
{
private:
    int N;
    float m;
    float s ;
    // Fastayrðing gagna:
    //  N er fjöldi talna í því úrtaki sem hefur verið lesið inn í eintak af Accumulator, N >= 0
    //  m er meðaltal talna í því úrtaki sem hefur verið lesið inn í eintak af Accumulator
    //  s er summa ferningsfrávika (frávik sérhvers gildis frá meðaltali, í öðru veldi), í því 
    //    úrtaki sem hefur verið lesið inn í eintak af Accumulator, s >= 0
public:
    // N: Accumulator a;
    // F: Ekkert
    // E: a er nýtt eintak af Accumulator, sem engar tölur hafa lesnar inn í. 
    //      Öll gögn í a hafa verið núllstillt, það er a.N = 0, a.m = 0.0 og a.s = 0.0
    Accumulator() {
        N = 0;
        m = 0.0;
        s = 0.0;
    }

    // N: a.addDataValue(x)
    // F: Ekkert
    // E: Búið er að bæta x í úrtakið a
    void addDataValue(float x)
    {
        N++;
        s = s + 1.0*(N-1)/N*(x-m)*(x-m);
        m = m + (x-m)/N;
    }

    // N: x = a.mean()
    // F: Ekkert
    // E: x inniheldur meðaltal talna í úrtakinu a
    float mean()
    {
        return m;
    }

    // N: x = a.var()
    // F: N > 1
    // E: x inniheldur dreifni talna í úrtakinu a
    float var()
    {
        return s/(N-1);
    }

    // N: x = a.stddev()
    // F: N > 1
    // E: x inniheldur staðalfrávik talna í úrtakinu a
    float stddev ( )
    {
        return sqrt(s/(N-1));
    }
};

int main(){
    // Minimum and maximum values of B, and number of steps. 
    // If Bsteps = 1, then only Bmin is used. 
    float B;
    float Bmin = 0.0;
    float Bmax = 1.0;
    int Bsteps = 1;
    // Minimum and maximum values of kT, and number of steps. 
    // If kTsteps = 1, then only kTmin is used. 
    float kT;
    float kTmin = 0.5;
    float kTmax = 5.0;
    int kTsteps = 1;
    // Minimum and maximum values of Q, and number of steps. 
    // If Qsteps = 1, then only Qmin is used. 
    float Q;
    float Qmin = -1.0;
    float Qmax = 1.0;
    int Qsteps = 1;

    srand(time(NULL)); // Seed GPU RNG
    float Emean;
    float Mmean;
    float Evar;
    float Mvar;

    
    char filename[20];
    sprintf(filename, "results.dat");
    FILE *fp;
    fp = fopen(filename, "w");
    for (int i=0;i<Bsteps;i++){ // B loop
        if (Bsteps>1){
            B = Bmin + i*(Bmax-Bmin)/(Bsteps-1);
        }
        else{
            B = Bmin;
        }
        for(int k=0; k<Qsteps; k++){ // Q loop
            if (Qsteps>1){
                Q = Qmin + k*(Qmax-Qmin)/(Qsteps-1);
            }
            else{
                Q = Qmin;
            }
            for(int j=0; j<kTsteps; j++){ // kTsteps
                if (kTsteps>1){
                    kT = kTmin + j*(kTmax-kTmin)/(kTsteps-1);
                }
                else{
                    kT = kTmin;
                }
                // printf("Performing calculation at B=%g, kT=%g, Q=%g\n", B, kT, Q);
                calc_cpu(B, kT, Q, &Emean, &Mmean, &Evar, &Mvar);
                fprintf(fp, "%.6e\t%.6e\t%.6e\t%.6e\t%.6e\t%.6e\t%.6e\n", B, kT, Q, Emean, Mmean, Evar, Mvar);
            }
        }
    }
    fclose(fp);
}

// U: calc_cpu(...)
// B: kT > 0, n => 0 
// A: The results of an ising simulation at magnetic field B and
//      temperature kT have been stored in Earr[n] (mean energy) 
//      and Marr[n] (mean magnetization)
void calc_cpu(float B, float kT, float Q, float *E_avg_out, float *M_avg_out, float *E_var_out, float *M_var_out){
    // Degbug things
    // Template:
    // cudaMemcpy( &buggy, dev_value, sizeof(float), cudaMemcpyDeviceToHost);
    // printf("%g\n",buggy);
    /*float buggy;*/
    /*float buggyvec[thread_num];*/

    // Create, allocate memory for and set lattice
    bool *dev_lattice;
    hipMalloc( (void**)&dev_lattice, grid_size*sizeof(bool) );
    set_lattice<<<1, thread_num>>>(dev_lattice);

    float *dev_dEvec;
    float *dev_dMvec;
    hipMalloc( (void**)&dev_dEvec, thread_num*sizeof(float) );
    hipMalloc( (void**)&dev_dMvec, thread_num*sizeof(float) );

    float *dev_Etot;
    float *dev_Mtot;
    /*float *dev_Eavg;*/
    /*float *dev_Mavg;*/
    hipMalloc( (void**)&dev_Etot, sizeof(float) );
    hipMalloc( (void**)&dev_Mtot, sizeof(float) );
    /*cudaMalloc( (void**)&dev_Eavg, sizeof(float) );*/
    /*cudaMalloc( (void**)&dev_Mavg, sizeof(float) );*/

    set_val<<<1,1>>>(dev_Etot, grid_size*(-2.0-2.0*Q-B));
    set_val<<<1,1>>>(dev_Mtot, grid_size);
    /*set_val<<<1,1>>>(dev_Eavg, 0.0);*/
    /*set_val<<<1,1>>>(dev_Mavg, 0.0);*/

    Accumulator energy;
    Accumulator magnet;

    float Etot;
    float Mtot;

    for (int j=0; j<iter; j++){
        reset_vec<<<1, thread_num>>>(dev_dEvec);
        reset_vec<<<1, thread_num>>>(dev_dMvec);

        iterate_grid<<<1, thread_num>>>(B, kT, Q, 0, dev_lattice, dev_dEvec, dev_dMvec, rand() );
        iterate_grid<<<1, thread_num>>>(B, kT, Q, 1, dev_lattice, dev_dEvec, dev_dMvec, rand() );

        vec_sum<<<1, thread_num>>>(dev_dEvec, dev_Etot);
        vec_sum<<<1, thread_num>>>(dev_dMvec, dev_Mtot);

        if (j>iterbal){
            hipMemcpy( &Etot, dev_Etot, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy( &Mtot, dev_Mtot, sizeof(float), hipMemcpyDeviceToHost);
            Etot = Etot/grid_size;
            Mtot = Mtot/grid_size;
            energy.addDataValue(Etot);
            magnet.addDataValue(Mtot);
        }
    }
    *E_avg_out = energy.mean();
    *M_avg_out = magnet.mean();
    *E_var_out = energy.var();
    *M_var_out = magnet.var();
    hipFree(dev_lattice);
    hipFree(dev_dEvec);
    hipFree(dev_dMvec);
    hipFree(dev_Etot);
    hipFree(dev_Mtot);

}

// U: set_lattice<<<1, thread_num>>>(dev_lattice);
// B: dev_lattice points to allocated device memory for grid_size bool numbers
// A: all elements of dev_lattice are set to 1
__global__ void set_lattice(bool *lattice){
    int tid = threadIdx.x;
    for (int i=tid;i<grid_size;i+=thread_num){
        lattice[i] = 1;
    }
}

// U: iterate_grid<<<1, thread_num>>>(...)
// B: 
// A: One ising iteration has been performed over a checkerboard. If round=0 it's over the white squares, if round=1 it's over 
//     the black squares. The change done by each thread has been added to d_E_vec[tid] and d_M_vec[tid]
__global__ void iterate_grid(float B, float kT, float Q, bool round, bool *dev_lattice, float *d_E_vec, float *d_M_vec, int seed){
    int tid=threadIdx.x;

    hiprandState_t state;
    hiprand_init(seed+tid, 0, 0, &state);

    int si;
    float ssum;
    float delta_E;
    float delta_M;
    float p;
    float r;
    int xi;
    int yi;
    for (int i=round+2*tid;i<grid_size;i+=2*thread_num){
        yi = i/grid_dim_x;
        if ((yi%2)==0){
            xi = i%grid_dim_x;
        }
        else{
            xi = grid_dim_x-i%grid_dim_x-1;
        }
        si = 2*dev_lattice[i]-1;
        ssum = 2*dev_lattice[indexMap(xi-1,yi)]
              +2*dev_lattice[indexMap(xi+1,yi)]
              -2
            +Q*2*dev_lattice[indexMap(xi,yi-1)]
            +Q*2*dev_lattice[indexMap(xi,yi+1)]
            -Q*2;
        delta_E = 2*si*(ssum+B);
        delta_M = -2*si;
        if (delta_E < 0){
            p = 1;
        }
        else{
            p = exp(-delta_E/kT);
        }
        r = hiprand_uniform(&state);
        if (r<p){ // Spin flip!
            d_E_vec[tid] += delta_E;
            d_M_vec[tid] += delta_M;
            dev_lattice[i] = !( dev_lattice[i] );
        }
    }
}

// U: reset_vec<<<1, thread_num>>>(dev_vec)
// B: dev_vec has been allocated device memory for thread_num float numbers
// A: All elements of dev_vec have been set as 0.0
__global__ void reset_vec(float *vec){
    vec[threadIdx.x] = 0.0;
}

// U: vec_sum<<<1, thread_num>>>(dev_vec, dev_result)
// B: dev_vec has length thread_num
// A: The sum of elements in dev_vec has been added to result
__global__ void vec_sum(float *vec, float *result){
    // Right multithread version (has to use threads)
    int tid = threadIdx.x;
    int offset = thread_num>>1;
    while (offset>0){
        if (tid < offset){
            vec[tid] += vec[tid+offset];
        }
        __syncthreads();
        offset=offset>>1;
    }
    if (tid==0){
        *result += vec[0];
    }
    
    // Right single thread version
    /*int tid = threadIdx.x;*/
    /*if (tid == 0){*/
        /*for (int i=1;i<thread_num;i++){*/
            /*vec[0] += vec[i];*/
        /*}*/
        /**result += vec[0];*/
    /*}*/
}

// U: set_val<<<1, 1>>>(variable, value)
// B: 
// A: *variable = value
__global__ void set_val(float *variable, float value){
    *variable = value;
}

// U: add_val<<<1, 1>>>(variable, addition)
// B:
// A: *variabe += *addition
__global__ void add_val(float *variable, float *addition){
    *variable += *addition;
}

// U: z = posMod(n,m)
// B: m > 0
// A: z = n%m if n>=0, z = n%m + m if n < 0
__device__ int posMod(int number, int modulus){
    int result = number%modulus;
    if (result<0){
        result +=modulus;
    }
    return result;
}

__device__ int indexMap(int xi, int yi){
    xi = posMod(xi,grid_dim_x);
    yi = posMod(yi,grid_dim_x);
    int i = yi*grid_dim_x;
    if (yi%2==0){
        i += xi;
    }
    else{
        i += grid_dim_x-xi-1;
    }
    return i;
}

